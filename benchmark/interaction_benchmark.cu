#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <chrono>
#include <iostream>
#include <string>

#include "benchmark.h"

// Host-side data preprocessing performance test
static void BM_HostPreprocess(benchmark::State& state) {
    try {
        const int N = state.range(0);
        std::cout << "\n[Starting] Host Preprocess benchmark [size: " << N << "]" << std::endl;

        InitCUDA();
        KernelMetrics metrics;
        metrics.size_kb = N * sizeof(float) / 1024.0;

        float* h_data = nullptr;
        auto start_total = std::chrono::high_resolution_clock::now();

        try {
            h_data = new float[N];

            std::cout << "[Running] Executing benchmark iterations..." << std::endl;
            CUDAEventTimer timer;

            for (auto _ : state) {
                timer.Start();
                for (int i = 0; i < N; i++) {
                    h_data[i] = float(i) * 0.5f;
                }
                timer.Stop();

                metrics.kernel_time = timer.ElapsedMillis();
                metrics.bandwidth = (N * sizeof(float)) / (metrics.kernel_time * 1e-3) / 1e9;

                state.SetIterationTime(metrics.kernel_time / 1000.0);
                state.counters["KernelTime_ms"] = metrics.kernel_time;
                state.counters["Bandwidth_GB/s"] = metrics.bandwidth;
                state.counters["GFLOPS"] = metrics.gflops;
                state.counters["Size_KB"] = metrics.size_kb;
            }

            delete[] h_data;

        } catch (...) {
            delete[] h_data;
            throw;
        }

        auto end_total = std::chrono::high_resolution_clock::now();
        metrics.total_time = std::chrono::duration<double, std::milli>(end_total - start_total).count();
        state.counters["TotalTime_ms"] = metrics.total_time;

        CleanupCUDA();
        std::cout << "[Completed] Host Preprocess benchmark" << std::endl;
    } catch (const std::exception& e) {
        state.SkipWithError(e.what());
    }
}

// Host-device data interaction performance test
static void BM_HostDeviceInteraction(benchmark::State& state) {
    try {
        const int N = state.range(0);
        std::cout << "\n[Starting] Host-Device Interaction benchmark [size: " << N << "]" << std::endl;

        InitCUDA();
        KernelMetrics metrics;
        metrics.size_kb = N * sizeof(float) * 2 / 1024.0;  // Input + output data

        float *h_data = nullptr, *d_data = nullptr;
        auto start_total = std::chrono::high_resolution_clock::now();

        try {
            CUDA_CHECK(hipHostMalloc(&h_data, N * sizeof(float)));
            CUDA_CHECK(hipMalloc(&d_data, N * sizeof(float)));

            std::cout << "[Running] Executing benchmark iterations..." << std::endl;
            CUDAEventTimer timer;

            for (auto _ : state) {
                timer.Start();

                // Host data preparation
                for (int i = 0; i < N; i++) {
                    h_data[i] = float(i);
                }

                // Transfer to device
                CUDA_CHECK(hipMemcpy(d_data, h_data, N * sizeof(float), hipMemcpyHostToDevice));

                // Read back from device
                CUDA_CHECK(hipMemcpy(h_data, d_data, N * sizeof(float), hipMemcpyDeviceToHost));

                // Host processing result
                float sum = 0.0f;
                for (int i = 0; i < N; i++) {
                    sum += h_data[i];
                }

                timer.Stop();

                metrics.kernel_time = timer.ElapsedMillis();
                metrics.bandwidth = (2.0 * N * sizeof(float)) / (metrics.kernel_time * 1e-3) / 1e9;

                state.SetIterationTime(metrics.kernel_time / 1000.0);
                state.counters["KernelTime_ms"] = metrics.kernel_time;
                state.counters["Bandwidth_GB/s"] = metrics.bandwidth;
                state.counters["GFLOPS"] = metrics.gflops;
                state.counters["Size_KB"] = metrics.size_kb;
            }

            CUDA_CHECK(hipHostFree(h_data));
            CUDA_CHECK(hipFree(d_data));
        } catch (...) {
            if (h_data) hipHostFree(h_data);
            if (d_data) hipFree(d_data);
            throw;
        }

        auto end_total = std::chrono::high_resolution_clock::now();
        metrics.total_time = std::chrono::duration<double, std::milli>(end_total - start_total).count();
        state.counters["TotalTime_ms"] = metrics.total_time;

        CleanupCUDA();
        std::cout << "[Completed] Host-Device Interaction benchmark" << std::endl;
    } catch (const std::exception& e) {
        state.SkipWithError(e.what());
    }
}

// Empty kernel for launch overhead test
__global__ void emptyKernel() {}

// Kernel launch overhead performance test
static void BM_KernelLaunchOverhead(benchmark::State& state) {
    try {
        std::cout << "\n[Starting] Kernel Launch Overhead benchmark" << std::endl;

        InitCUDA();
        KernelMetrics metrics;
        metrics.size_kb = 0;  // No data transfer

        auto start_total = std::chrono::high_resolution_clock::now();
        CUDAEventTimer timer;

        std::cout << "[Running] Executing benchmark iterations..." << std::endl;
        for (auto _ : state) {
            timer.Start();
            emptyKernel<<<1, 1>>>();
            CUDA_CHECK(hipDeviceSynchronize());
            timer.Stop();

            metrics.kernel_time = timer.ElapsedMillis();

            state.SetIterationTime(metrics.kernel_time / 1000.0);
            state.counters["KernelTime_ms"] = metrics.kernel_time;
            state.counters["Bandwidth_GB/s"] = metrics.bandwidth;
            state.counters["GFLOPS"] = metrics.gflops;
            state.counters["Size_KB"] = metrics.size_kb;
        }

        auto end_total = std::chrono::high_resolution_clock::now();
        metrics.total_time = std::chrono::duration<double, std::milli>(end_total - start_total).count();
        state.counters["TotalTime_ms"] = metrics.total_time;

        CleanupCUDA();
        std::cout << "[Completed] Kernel Launch Overhead benchmark" << std::endl;
    } catch (const std::exception& e) {
        state.SkipWithError(e.what());
    }
}

// Register host preprocess benchmark
BENCHMARK(BM_HostPreprocess)
    ->RangeMultiplier(2)
    ->Range(1 << 8, 1 << 10)
    ->UseManualTime()
    ->Unit(benchmark::kMicrosecond)
    ->Repetitions(2);

// Register host-device interaction benchmark
BENCHMARK(BM_HostDeviceInteraction)
    ->RangeMultiplier(2)
    ->Range(1 << 8, 1 << 10)
    ->UseManualTime()
    ->Unit(benchmark::kMicrosecond)
    ->Repetitions(2);

// Register kernel launch overhead benchmark
BENCHMARK(BM_KernelLaunchOverhead)
    ->UseManualTime()
    ->Unit(benchmark::kMicrosecond)
    ->Repetitions(2);

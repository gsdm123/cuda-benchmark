#include <hip/hip_runtime.h>

#include <chrono>
#include <iostream>
#include <string>

#include "benchmark.h"

// Device management performance test
static void BM_CudaSetDevice(benchmark::State& state) {
    auto start_total = std::chrono::high_resolution_clock::now();

    std::cout << "\n[Starting] CudaSetDevice benchmark" << std::endl;

    // Initialize CUDA and metrics
    InitCUDA();
    KernelMetrics metrics;
    metrics.size_kb = 0;  // No data transfer

    // Benchmark iterations
    std::cout << "[Running] Executing benchmark iterations..." << std::endl;
    CUDAEventTimer timer;

    for (auto _ : state) {
        timer.Start();
        CUDA_CHECK(hipSetDevice(0));
        timer.Stop();

        metrics.kernel_time = timer.ElapsedMillis();

        state.SetIterationTime(metrics.kernel_time / 1000.0);
        state.counters["KernelTime_ms"] = metrics.kernel_time;
        state.counters["Bandwidth_GB/s"] = metrics.bandwidth;
        state.counters["GFLOPS"] = metrics.gflops;
        state.counters["Size_KB"] = metrics.size_kb;
    }

    auto end_total = std::chrono::high_resolution_clock::now();
    metrics.total_time = std::chrono::duration<double, std::milli>(end_total - start_total).count();
    state.counters["TotalTime_ms"] = metrics.total_time;

    // Cleanup
    CleanupCUDA();
    std::cout << "[Completed] CudaSetDevice benchmark" << std::endl;
}

// Get device properties performance test
static void BM_CudaGetDeviceProperties(benchmark::State& state) {
    auto start_total = std::chrono::high_resolution_clock::now();

    std::cout << "\n[Starting] CudaGetDeviceProperties benchmark" << std::endl;

    // Initialize CUDA and metrics
    InitCUDA();
    KernelMetrics metrics;
    metrics.size_kb = sizeof(hipDeviceProp_t) / 1024.0;

    // Benchmark iterations
    std::cout << "[Running] Executing benchmark iterations..." << std::endl;
    CUDAEventTimer timer;
    hipDeviceProp_t prop;

    for (auto _ : state) {
        timer.Start();
        CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
        timer.Stop();

        metrics.kernel_time = timer.ElapsedMillis();

        state.SetIterationTime(metrics.kernel_time / 1000.0);
        state.counters["KernelTime_ms"] = metrics.kernel_time;
        state.counters["Bandwidth_GB/s"] = metrics.bandwidth;
        state.counters["GFLOPS"] = metrics.gflops;
        state.counters["Size_KB"] = metrics.size_kb;
    }

    auto end_total = std::chrono::high_resolution_clock::now();
    metrics.total_time = std::chrono::duration<double, std::milli>(end_total - start_total).count();
    state.counters["TotalTime_ms"] = metrics.total_time;

    // Cleanup
    CleanupCUDA();
    std::cout << "[Completed] CudaGetDeviceProperties benchmark" << std::endl;
}

// Context synchronization performance test
static void BM_CudaDeviceSynchronize(benchmark::State& state) {
    auto start_total = std::chrono::high_resolution_clock::now();

    std::cout << "\n[Starting] CudaDeviceSynchronize benchmark" << std::endl;

    // Initialize CUDA and metrics
    InitCUDA();
    KernelMetrics metrics;
    metrics.size_kb = 0;  // No data transfer

    // Benchmark iterations
    std::cout << "[Running] Executing benchmark iterations..." << std::endl;
    CUDAEventTimer timer;

    for (auto _ : state) {
        timer.Start();
        CUDA_CHECK(hipDeviceSynchronize());
        timer.Stop();

        metrics.kernel_time = timer.ElapsedMillis();

        state.SetIterationTime(metrics.kernel_time / 1000.0);
        state.counters["KernelTime_ms"] = metrics.kernel_time;
        state.counters["Bandwidth_GB/s"] = metrics.bandwidth;
        state.counters["GFLOPS"] = metrics.gflops;
        state.counters["Size_KB"] = metrics.size_kb;
    }

    auto end_total = std::chrono::high_resolution_clock::now();
    metrics.total_time = std::chrono::duration<double, std::milli>(end_total - start_total).count();
    state.counters["TotalTime_ms"] = metrics.total_time;

    // Cleanup
    CleanupCUDA();
    std::cout << "[Completed] CudaDeviceSynchronize benchmark" << std::endl;
}

// Event management performance test
static void BM_CudaEventCreateDestroy(benchmark::State& state) {
    auto start_total = std::chrono::high_resolution_clock::now();

    std::cout << "\n[Starting] CudaEventCreateDestroy benchmark" << std::endl;

    // Initialize CUDA and metrics
    InitCUDA();
    KernelMetrics metrics;
    metrics.size_kb = 0;  // No data transfer

    // Benchmark iterations
    std::cout << "[Running] Executing benchmark iterations..." << std::endl;
    CUDAEventTimer timer;

    for (auto _ : state) {
        hipEvent_t event;
        timer.Start();
        CUDA_CHECK(hipEventCreate(&event));
        CUDA_CHECK(hipEventDestroy(event));
        timer.Stop();

        metrics.kernel_time = timer.ElapsedMillis();

        state.SetIterationTime(metrics.kernel_time / 1000.0);
        state.counters["KernelTime_ms"] = metrics.kernel_time;
        state.counters["Bandwidth_GB/s"] = metrics.bandwidth;
        state.counters["GFLOPS"] = metrics.gflops;
        state.counters["Size_KB"] = metrics.size_kb;
    }

    auto end_total = std::chrono::high_resolution_clock::now();
    metrics.total_time = std::chrono::duration<double, std::milli>(end_total - start_total).count();
    state.counters["TotalTime_ms"] = metrics.total_time;

    // Cleanup
    CleanupCUDA();
    std::cout << "[Completed] CudaEventCreateDestroy benchmark" << std::endl;
}

// Register setDevice benchmark
BENCHMARK(BM_CudaSetDevice)->UseManualTime()->Unit(benchmark::kMicrosecond)->Repetitions(2);

// Register getDeviceProperties benchmark
BENCHMARK(BM_CudaGetDeviceProperties)
    ->UseManualTime()
    ->Unit(benchmark::kMicrosecond)
    ->Repetitions(2);

// Register deviceSynchronize benchmark
BENCHMARK(BM_CudaDeviceSynchronize)->UseManualTime()->Unit(benchmark::kMicrosecond)->Repetitions(2);

// Register eventCreateDestroy benchmark
BENCHMARK(BM_CudaEventCreateDestroy)
    ->UseManualTime()
    ->Unit(benchmark::kMicrosecond)
    ->Repetitions(2);

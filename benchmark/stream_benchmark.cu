#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <chrono>
#include <iostream>
#include <string>

#include "benchmark.h"

// Simple vector addition kernel
__global__ void kernelStreamSample(const float* a, const float* b, float* c, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

// Stream creation and destruction performance test
static void BM_StreamCreateDestroy(benchmark::State& state) {
    auto start_total = std::chrono::high_resolution_clock::now();

    std::cout << "\n[Starting] Stream Create/Destroy benchmark" << std::endl;

    // Initialize CUDA and metrics
    InitCUDA();
    KernelMetrics metrics;
    metrics.size_kb = 0;  // No data transfer

    // Benchmark iterations
    std::cout << "[Running] Executing benchmark iterations..." << std::endl;
    CUDAEventTimer timer;

    for (auto _ : state) {
        hipStream_t stream;
        timer.Start();
        CUDA_CHECK(hipStreamCreate(&stream));
        CUDA_CHECK(hipStreamDestroy(stream));
        timer.Stop();

        metrics.kernel_time = timer.ElapsedMillis();

        state.SetIterationTime(metrics.kernel_time / 1000.0);
        state.counters["KernelTime_ms"] = metrics.kernel_time;
        state.counters["Bandwidth_GB/s"] = metrics.bandwidth;
        state.counters["GFLOPS"] = metrics.gflops;
        state.counters["Size_KB"] = metrics.size_kb;
    }

    auto end_total = std::chrono::high_resolution_clock::now();
    metrics.total_time = std::chrono::duration<double, std::milli>(end_total - start_total).count();
    state.counters["TotalTime_ms"] = metrics.total_time;

    // Cleanup
    CleanupCUDA();
    std::cout << "[Completed] Stream Create/Destroy benchmark" << std::endl;
}

// Stream synchronization performance test
static void BM_StreamSynchronize(benchmark::State& state) {
    auto start_total = std::chrono::high_resolution_clock::now();
    std::cout << "\n[Starting] Stream Synchronize benchmark" << std::endl;

    // Initialize CUDA and metrics
    InitCUDA();
    KernelMetrics metrics;
    metrics.size_kb = 0;  // No data transfer

    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    // Benchmark iterations
    std::cout << "[Running] Executing benchmark iterations..." << std::endl;
    CUDAEventTimer timer;

    for (auto _ : state) {
        timer.Start();
        CUDA_CHECK(hipStreamSynchronize(stream));
        timer.Stop();

        metrics.kernel_time = timer.ElapsedMillis();

        state.SetIterationTime(metrics.kernel_time / 1000.0);
        state.counters["KernelTime_ms"] = metrics.kernel_time;
        state.counters["Bandwidth_GB/s"] = metrics.bandwidth;
        state.counters["GFLOPS"] = metrics.gflops;
        state.counters["Size_KB"] = metrics.size_kb;
    }

    auto end_total = std::chrono::high_resolution_clock::now();
    metrics.total_time = std::chrono::duration<double, std::milli>(end_total - start_total).count();
    state.counters["TotalTime_ms"] = metrics.total_time;

    // Cleanup
    CUDA_CHECK(hipStreamDestroy(stream));
    CleanupCUDA();
    std::cout << "[Completed] Stream Synchronize benchmark" << std::endl;
}

// Multi-stream concurrency execution performance test
static void BM_StreamConcurrency(benchmark::State& state) {
    auto start_total = std::chrono::high_resolution_clock::now();

    const int N = state.range(0);
    const int numStreams = 4;
    const int streamSize = N / numStreams;

    std::cout << "\n[Starting] Stream Concurrency benchmark [size: " << N << "]" << std::endl;

    // Initialize CUDA and metrics
    InitCUDA();
    KernelMetrics metrics;
    metrics.size_kb = N * sizeof(float) * 3 / 1024.0;  // Input + output data

    // Allocate and initialize data
    float *h_a = nullptr, *h_b = nullptr, *h_c = nullptr;
    float *d_a = nullptr, *d_b = nullptr, *d_c = nullptr;
    // Memory allocation and data initialization
    CUDA_CHECK(hipHostMalloc(&h_a, N * sizeof(float)));
    CUDA_CHECK(hipHostMalloc(&h_b, N * sizeof(float)));
    CUDA_CHECK(hipHostMalloc(&h_c, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_a, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_b, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_c, N * sizeof(float)));

    std::fill_n(h_a, N, 1.0f);
    std::fill_n(h_b, N, 2.0f);

    // Create streams
    hipStream_t streams[numStreams];
    for (int i = 0; i < numStreams; ++i) {
        CUDA_CHECK(hipStreamCreate(&streams[i]));
    }

    // Configure kernel
    int blockSize = 256;
    int streamBlocks = (streamSize + blockSize - 1) / blockSize;

    // Warm up
    std::cout << "[Warmup] Running warmup iteration..." << std::endl;
    for (int i = 0; i < numStreams; ++i) {
        int offset = i * streamSize;
        kernelStreamSample<<<streamBlocks, blockSize, 0, streams[i]>>>(d_a + offset, d_b + offset,
                                                                       d_c + offset, streamSize);
    }
    CUDA_CHECK(hipDeviceSynchronize());

    // Benchmark iterations
    std::cout << "[Running] Executing benchmark iterations..." << std::endl;
    CUDAEventTimer timer;

    for (auto _ : state) {
        timer.Start();
        // Launch operations in multiple streams
        for (int i = 0; i < numStreams; ++i) {
            int offset = i * streamSize;
            CUDA_CHECK(hipMemcpyAsync(d_a + offset, h_a + offset, streamSize * sizeof(float),
                                       hipMemcpyHostToDevice, streams[i]));
            CUDA_CHECK(hipMemcpyAsync(d_b + offset, h_b + offset, streamSize * sizeof(float),
                                       hipMemcpyHostToDevice, streams[i]));
            kernelStreamSample<<<streamBlocks, blockSize, 0, streams[i]>>>(
                d_a + offset, d_b + offset, d_c + offset, streamSize);
            CUDA_CHECK(hipMemcpyAsync(h_c + offset, d_c + offset, streamSize * sizeof(float),
                                       hipMemcpyDeviceToHost, streams[i]));
        }

        // Wait for all streams
        for (int i = 0; i < numStreams; ++i) {
            CUDA_CHECK(hipStreamSynchronize(streams[i]));
        }
        timer.Stop();

        metrics.kernel_time = timer.ElapsedMillis();
        metrics.bandwidth = (metrics.size_kb * 1024) / (metrics.kernel_time * 1e-3) / 1e9;
        metrics.gflops = (1.0 * N) / (metrics.kernel_time * 1e-3) / 1e9;

        state.SetIterationTime(metrics.kernel_time / 1000.0);
        state.counters["KernelTime_ms"] = metrics.kernel_time;
        state.counters["Bandwidth_GB/s"] = metrics.bandwidth;
        state.counters["GFLOPS"] = metrics.gflops;
        state.counters["Size_KB"] = metrics.size_kb;
    }

    auto end_total = std::chrono::high_resolution_clock::now();
    metrics.total_time = std::chrono::duration<double, std::milli>(end_total - start_total).count();
    state.counters["TotalTime_ms"] = metrics.total_time;

    // Cleanup
    CUDA_CHECK(hipHostFree(h_a));
    CUDA_CHECK(hipHostFree(h_b));
    CUDA_CHECK(hipHostFree(h_c));
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_c));
    for (int i = 0; i < numStreams; ++i) {
        CUDA_CHECK(hipStreamDestroy(streams[i]));
    }
    CleanupCUDA();
    std::cout << "[Completed] Stream Concurrency benchmark" << std::endl;
}

// Register stream creation and destruction benchmark
BENCHMARK(BM_StreamCreateDestroy)->UseManualTime()->Unit(benchmark::kMicrosecond)->Repetitions(2);

// Register stream synchronization benchmark
BENCHMARK(BM_StreamSynchronize)->UseManualTime()->Unit(benchmark::kMicrosecond)->Repetitions(2);

// Register multi-stream concurrency benchmark
BENCHMARK(BM_StreamConcurrency)
    ->RangeMultiplier(2)
    ->Range(1 << 8, 1 << 10)
    ->UseManualTime()
    ->Unit(benchmark::kMicrosecond)
    ->Repetitions(2);
